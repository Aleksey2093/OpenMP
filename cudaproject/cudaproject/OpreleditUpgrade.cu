#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "CudaInfo.cuh"
#include "hip/hip_runtime.h"
#include ""

#define SIZE 3

__host__
void matrix_view(float(*array)[SIZE], char *q) {                  //����� ������� �� �����
	int i, j;
	for (i = 0; i<SIZE; i++) {
		for (j = 0; j<SIZE; j++)
			printf(q, array[i][j]);
		puts("");
	}
	puts("");
}

__host__
void matrix_rand(float(*array)[SIZE]) {               //���������� ��������� ������� �� �������
	int i, j;
	for (i = 0; i<SIZE; i++)
		for (j = 0; j<SIZE; j++)
			array[i][j] = 1 + rand() % 16;
}

__device__
void subrow(float(*array)[SIZE], int m, int n, float k) {         //��������� ����� �� ����������
	int g = blockIdx.y*blockDim.y + threadIdx.y;
	if (g<SIZE)
		array[m][g] -= k*array[n][g];
}

__global__
void determinant(float(*mtx)[SIZE]) {             //����: ���������� � ������������ ����
	int i = i = blockIdx.x*blockDim.x + threadIdx.x;
	int j;
	float coeff;
	for (j = 0; j<SIZE - 1; j++) {
		if (!mtx[j][j]) subrow(mtx, j, SIZE - 1, 3);
		if (i >= j && i<SIZE - 1) {
			coeff = mtx[i + 1][j] / mtx[j][j];
			subrow(mtx, i + 1, j, coeff);
		}
		__syncthreads();
	}
}

int CudaInfo::OpredelitUpgrade(void)
{
	int i;
	float mtx_h[SIZE][SIZE], (*mtx_d)[SIZE];
	long double det;
	hipMalloc((void **)&mtx_d, sizeof(float)*SIZE*SIZE);          //��������� ������ �� ����������
	puts("�������� �������\n");
	matrix_rand(mtx_h);
	hipMemcpy(mtx_d, mtx_h, sizeof(float)*SIZE*SIZE,       //����������� ������� � ������ ����������
		hipMemcpyHostToDevice);
	matrix_view(mtx_h, "| %.0f ");
	dim3 threadsPerBlock(16, 16);
	dim3 numBlock(SIZE / threadsPerBlock.x, SIZE / threadsPerBlock.y);
	determinant << <numBlock, threadsPerBlock >> >(mtx_d);                      //����� ����
	hipDeviceSynchronize();
	hipMemcpy(mtx_h, mtx_d, sizeof(float)*SIZE*SIZE,       //����������� ������� �� ������ ����������
		hipMemcpyDeviceToHost);
	puts("��e�������� ���\n");
	matrix_view(mtx_h, "| %.8f ");
	det = 1;
	for (i = 0; i<SIZE; i++) {
		printf("%.8f\n", mtx_h[i][i]);
		det *= mtx_h[i][i];
	}
	printf("det=%.0Lf\n ", det);
	hipFree(mtx_d);
	getchar();
	return 0;
}