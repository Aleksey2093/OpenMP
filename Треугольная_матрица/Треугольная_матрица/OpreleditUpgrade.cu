#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "CudaInfo.cuh"
#include "hip/hip_runtime.h"
#include ""

__host__
void matrix_view(float(*array)[SIZE], char *q) {                  //����� ������� �� �����
	int i, j;
	for (i = 0; i<SIZE; i++) {
		for (j = 0; j<SIZE; j++)
			printf(q, array[i][j]);
		puts("");
	}
	puts("");
}

__host__
void matrix_rand(float(*array)[SIZE], double **matrix, int n) {               //���������� ��������� ������� �� �������
	int i, j;
	for (i = 0; i < SIZE; i++)
		for (j = 0; j < SIZE; j++)
			array[i][j] = /*matrix[i][j]*/ 1 + rand() % 16;
}

__device__
void subrow(float(*array)[SIZE], int m, int n, float k) {         //��������� ����� �� ����������
	int g = blockIdx.y*blockDim.y + threadIdx.y;
	if (g<SIZE)
		array[m][g] -= k*array[n][g];
}

__global__
void determinant(float(*mtx)[SIZE]) {             //����: ���������� � ������������ ����
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j;
	float coeff;
	for (j = 0; j<SIZE - 1; j++) {
		if (!mtx[j][j]) subrow(mtx, j, SIZE - 1, 3);
		if (i >= j && i<SIZE - 1) {
			coeff = mtx[i + 1][j] / mtx[j][j];
			subrow(mtx, i + 1, j, coeff);
		}
		__syncthreads();
	}
}

int CudaInfo::OpredelitUpgrade(float mtx_h[SIZE][SIZE], int n)//(double **matrix, int n)
{
	int i;
	float /*mtx_h[SIZE][SIZE],*/ (*mtx_d)[SIZE];
	long double det;
	hipMalloc((void **)&mtx_d, sizeof(float)*SIZE*SIZE);          //��������� ����� �� ����������
	puts("�������� �������\n");
//	matrix_rand(mtx_h,matrix,n);
	time_t t1 = clock();
	hipMemcpy(mtx_d, mtx_h, sizeof(float)*SIZE*SIZE,       //����������� ������� � ����� ����������
		hipMemcpyHostToDevice);
	if (SIZE<10)
		matrix_view(mtx_h, "| %.0f ");
	dim3 threadsPerBlock(SIZE, SIZE, 1);
	dim3 numBlock(SIZE / threadsPerBlock.x, SIZE / threadsPerBlock.y, 1);
	determinant << <numBlock, threadsPerBlock >> >(mtx_d);                      //����� ����
	hipDeviceSynchronize();
	hipMemcpy(mtx_h, mtx_d, sizeof(float)*SIZE*SIZE,       //����������� ������� �� ������ ����������
		hipMemcpyDeviceToHost);
	puts("����������� ���\n");
	if (SIZE<10)
	matrix_view(mtx_h, "| %.8f ");
	det = 1;
	for (i = 0; i<SIZE; i++) {
		//printf("%.8f\n", mtx_h[i][i]);
		det *= mtx_h[i][i];
	}
	time_t t2 = clock();
	double timerun = ((t2 - t1) / CLOCKS_PER_SEC);
	printf("time work=%f\n",timerun);
	printf("det=%.0Lf\n ", det);
	hipFree(mtx_d);
	return 0;
}