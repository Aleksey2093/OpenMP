#include "hip/hip_runtime.h"
#include "Main.h"
#include "hip/hip_runtime.h"
#include ""

__device__ int Row;
__device__ int Col;

__global__ void determ()
{

}

__global__ void swaprow(float *Matrix, int row1, int row2, int n)
{
	int id = blockIdx.x + row1*n;
	int id2 = blockIdx.x + row2*n;
	float t = Matrix[id];
	Matrix[id] = Matrix[id2];
	Matrix[id2] = t;
}


__global__ void proizrow(float *Matrix, int row1, int n, float val)
{
	__shared__ float koe[SIZE];

}

/*for (int j = i + 1; j < SIZE; j++)
{
float k = Matrixline[j*SIZE + i] / val;
Matrixline[j*SIZE + i] = 0;
for (int c = i + 1; c < SIZE; c++)
Matrixline[j*SIZE + c] = Matrixline[j*SIZE + c] - Matrixline[i*SIZE + c] * k;
}*/

__global__ void seamax(float *Matrix, int rowstart, int colstart, int n, int g, int th)
{
	/*		
	for (int j = i + 1; j < SIZE; j++) //����� ������������� ��������
		{
			double tmp = fabs(Matrix[j][i]);
			if (tmp > maxValue)
			{
				maxN = j;
				maxValue = tmp;
			}
		}*/
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	__shared__ float maxblock[(SIZE + 127) / 128];// = new float[g]; //������ ���������� �� ������
	maxblock[threadIdx.x + blockDim.x * blockIdx.x] = Matrix[rowstart*n + colstart];
	for (int i = 0; i < th; i++)
	{

	}
}

double** Main::getMatrixFromCuda(double** Matrix)
{
	int n = SIZE*SIZE;
	float *dev_Matrix, *Matrixline = new float[n];
	float* row1, rowc1, row2, rowc2; float max;
	int g = (SIZE + 127) / 128, th = 128;
	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			Matrixline[i*SIZE + j] = Matrix[i][j];
		}
	}
	/*hipMalloc((void**)&row1, SIZE*sizeof(float));
	hipMalloc((void**)&row2, SIZE*sizeof(float));
	hipMalloc((void**)&max, sizeof(float));*/
	hipMalloc((void**)&dev_Matrix, n*sizeof(float));
	hipMemcpy(dev_Matrix, Matrixline, n*sizeof(float), hipMemcpyHostToDevice);

	int maxN; float maxValue;
	printf("\n");
	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			printf("%.2f\t",Matrixline[i*SIZE + j]);
		}
		printf("\n");
	}
	printf("\n");
	for (int i = 0; i < SIZE-1; i++)
	{
		//seamax << <g, th >> >(dev_Matrix, i, i + 1, SIZE, g, th);
//#pragma omp parallel
		{
			maxN = i;
			maxValue = fabs(Matrixline[i*SIZE+i]);
			for (int j = i + 1; j < SIZE; j++) //����� ������������� ��������
			{
				float tmp = fabs(Matrixline[j*SIZE+i]);
				if (tmp > maxValue)
				{
					maxN = j;
					maxValue = tmp;
				}
			}
			if (maxN > i)
			{
				hipMemcpy(dev_Matrix, Matrixline, n*sizeof(float), hipMemcpyHostToDevice);
				swaprow<<<SIZE,1>>>(dev_Matrix, i, maxN, SIZE);
				hipMemcpy(Matrixline, dev_Matrix, n*sizeof(float), hipMemcpyDeviceToHost);
			}
			else if (maxValue == 0)
			{
				printf("null in matrix\n");
				//return NULL;
			}
			float val = Matrixline[i *SIZE + i];



			/*for (int j = i + 1; j < SIZE; j++)
			{
				float k = Matrixline[j*SIZE + i] / val;
				Matrixline[j*SIZE + i] = 0;
				for (int c = i + 1; c < SIZE; c++)
					Matrixline[j*SIZE + c] = Matrixline[j*SIZE + c] - Matrixline[i*SIZE + c] * k;
			}*/
			for (int i = 0; i < SIZE; i++)
			{
				for (int j = 0; j < SIZE; j++)
				{
					printf("%.2f\t", Matrixline[i*SIZE + j]);
				}
				printf("\n");
			}
			printf("\n");
		}
	}

	/*hipMalloc((void**)&dev_Matrix, n*sizeof(float));
	hipMemcpy(dev_Matrix, Matrix, n*sizeof(float), hipMemcpyHostToDevice);
	dim3 grid(n, 1);
	add << <SIZE, 1 >> >(dev_Matrix);
	hipMemcpy(Matrix, dev_Matrix, n*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(dev_Matrix);*/

	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			printf("%.2f\t", Matrixline[i*SIZE + j]);
		}
		printf("\n");
	}

	return Matrix;
}