#include "hip/hip_runtime.h"
#include "Main.h"
#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime_api.h>

__global__ void determ()
{

}

__global__ void swaprow(float *Matrix, int row1, int row2, int n)
{
	float t;
	int id = blockIdx.x + row1*n;
	int id2 = blockIdx.x + row2*n;
	Matrix[id] = Matrix[id2];
	Matrix[id2] = t;
}


__global__ void proizrow(float *Matrix, int i, int j, int n, float val)
{
	/*__shared__ float koe[SIZE];
	int id = blockIdx.x + i*n;
	float row1[SIZE];
	if (blockIdx.x < SIZE) //������ ������� ����� ������ ��� ������ ���� �� ��� � �������� ������ ����������
	{
		row1[blockIdx.x] = Matrix[id];
		//printf("%.1f\t", row1[blockIdx.x]);
	}
	if (blockIdx.x >= SIZE)
	{
		for (int p = 0; p < SIZE; p++)
		{
			Matrix[blockIdx.x + j*n] = Matrix[blockIdx.x + j*n] + row1[p];
		}
	}*/
	int id = blockIdx.x + i*n;
	int id2 = blockIdx.x + j*n;
	float t = Matrix[id2] - Matrix[id];// *Matrix[i*SIZE + j] / val;
	if (id2 == i*SIZE + j)
	{
		Matrix[id2] = 0;
	}
	printf("(%.1f)", Matrix[i*SIZE + j]);
	Matrix[id2] = t;
}

/*for (int j = i + 1; j < SIZE; j++)
{
float k = Matrixline[j*SIZE + i] / val;
Matrixline[j*SIZE + i] = 0;
for (int c = i + 1; c < SIZE; c++)
Matrixline[j*SIZE + c] = Matrixline[j*SIZE + c] - Matrixline[i*SIZE + c] * k;
}*/

__global__ void seamax(float *Matrix, int rowstart, int colstart, int n, int g, int th)
{
	/*		
	for (int j = i + 1; j < SIZE; j++) //����� ������������� ��������
		{
			double tmp = fabs(Matrix[j][i]);
			if (tmp > maxValue)
			{
				maxN = j;
				maxValue = tmp;
			}
		}*/
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	__shared__ float maxblock[(SIZE + 127) / 128]; //������ ���������� �� ������
	maxblock[threadIdx.x + blockDim.x * blockIdx.x] = Matrix[rowstart*n + colstart];
	for (int i = 0; i < th; i++)
	{

	}
}

double** Main::getMatrixFromCuda(double** Matrix)
{
	int n = SIZE*SIZE;
	float *dev_Matrix, *Matrixline = new float[n];
	int g = (SIZE + 127) / 128, th = 128;
	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			Matrixline[i*SIZE + j] = Matrix[i][j];
		}
	}
	hipMalloc((void**)&dev_Matrix, n*sizeof(float));
	hipMemcpy(dev_Matrix, Matrixline, n*sizeof(float), hipMemcpyHostToDevice);

	int maxN; float maxValue;
	printf("\n");
	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			printf("%.2f\t",Matrixline[i*SIZE + j]);
		}
		printf("\n");
	}
	printf("\n");
	for (int i = 0; i < SIZE-1; i++)
	{
		//seamax << <g, th >> >(dev_Matrix, i, i + 1, SIZE, g, th);
//#pragma omp parallel
		{
			maxN = i;
			maxValue = fabs(Matrixline[i*SIZE+i]);
			for (int j = i + 1; j < SIZE; j++) //����� ������������� ��������
			{
				float tmp = fabs(Matrixline[j*SIZE+i]);
				if (tmp > maxValue)
				{
					maxN = j;
					maxValue = tmp;
				}
			}
			if (maxN > i)
			{
				hipMemcpy(dev_Matrix, Matrixline, n*sizeof(float), hipMemcpyHostToDevice);
				swaprow<<<SIZE,1>>>(dev_Matrix, i, maxN, SIZE);
				hipMemcpy(Matrixline, dev_Matrix, n*sizeof(float), hipMemcpyDeviceToHost);
			}
			else if (maxValue == 0)
			{
				printf("null in matrix\n");
			}
			float val = Matrixline[i *SIZE + i];

			//hipMemcpy(dev_Matrix, Matrixline, n*sizeof(float), hipMemcpyHostToDevice);
			for (int j = i + 1; j < SIZE; j++)
			{
				//float k = Matrixline[j * SIZE + i] / val;
				hipMemcpy(dev_Matrix, Matrixline, n*sizeof(float), hipMemcpyHostToDevice);
				proizrow << <SIZE, 1 >> >(dev_Matrix, i, j, SIZE, val);
				hipMemcpy(Matrixline, dev_Matrix, n*sizeof(float), hipMemcpyDeviceToHost);
			}
			//hipMemcpy(Matrixline, dev_Matrix, n*sizeof(float), hipMemcpyDeviceToHost);

			for (int i = 0; i < SIZE; i++)
			{
				for (int j = 0; j < SIZE; j++)
				{
					printf("%.2f\t", Matrixline[i*SIZE + j]);
				}
				printf("\n");
			}
			printf("\n");
			//__global__ void proizrow(float *Matrix, int i, int j, int n, float val)
			/*for (int j = i + 1; j < SIZE; j++)
			{
				float k = Matrixline[j*SIZE + i] / val;
				Matrixline[j*SIZE + i] = 0;
				for (int c = i + 1; c < SIZE; c++)
					Matrixline[j*SIZE + c] = Matrixline[j*SIZE + c] - Matrixline[i*SIZE + c] * k;
			}*/
		}
	}

	/*hipMalloc((void**)&dev_Matrix, n*sizeof(float));
	hipMemcpy(dev_Matrix, Matrix, n*sizeof(float), hipMemcpyHostToDevice);
	dim3 grid(n, 1);
	add << <SIZE, 1 >> >(dev_Matrix);
	hipMemcpy(Matrix, dev_Matrix, n*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(dev_Matrix);*/

	for (int i = 0; i < SIZE; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			printf("%.2f\t", Matrixline[i*SIZE + j]);
		}
		printf("\n");
	}

	return Matrix;
}