#include "hip/hip_runtime.h"
#include "Main.h"
#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime_api.h>

__device__ int maxNcuda;
__device__ float maxValueCuda;

__global__ void determ()
{

}

__global__ void swaprow(float *Matrix, int row1, int row2, int n)
{
	int id = blockIdx.x + row1*n;
	int id2 = blockIdx.x + row2*n;
	__shared__ float tmp[SIZE];
	tmp[blockIdx.x] = Matrix[id];
	Matrix[id] = Matrix[id2];
	Matrix[id2] = tmp[blockIdx.x];
}

__global__ void proizrow(float *Matrix, int i, int j, int n)
{
	int id = i*n + threadIdx.x; //������ ������
	int id2 = blockIdx.x*blockDim.x /*+ j*n*/ + threadIdx.x; //�����������
	if (id2 < n*n)
	{
		float jim = Matrix[j*n + i];
		float val = Matrix[i*n + i];
		__syncthreads();
		if (id2 == j*n + i)
		{
			Matrix[id2] = 0;
		}
		__syncthreads();
		if (id2 > j * n + i)
		{
			float t = Matrix[id2] - Matrix[id] * (jim / val);
			Matrix[id2] = t;
		}
	}
}

__global__ void seamax(float *Matrix, int row1, int n)
{

}

double** Main::getMatrixFromCuda(double** Matrix, Main *cu)
{
	int n = SIZE;
	double end, start;
	float *dev_Matrix, *Matrixline = new float[n*n];
	int g = (n*n + 127) / 128, th = 9;
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			Matrixline[i*n + j] = Matrix[i][j];
		}
	}
	hipMalloc((void**)&dev_Matrix, n*n*sizeof(float));
	hipMemcpy(dev_Matrix, Matrixline, n*n*sizeof(float), hipMemcpyHostToDevice);

	int maxN; float maxValue;
	
	if (n < 10)
	{
		printf("\n");
		for (int i = 0; i < n; i++)
		{
			for (int j = 0; j < n; j++)
			{
				printf("%.2f\t", Matrixline[i*n + j]);
			}
			printf("\n");
		}
		printf("\n");
	}
	start = omp_get_wtime();
	for (int i = 0; i < n - 1; i++)
	{
#pragma omp parallel
		{
#pragma omp sections
			{
#pragma omp section
				{
					maxN = i;
				}
#pragma omp section
				{
					maxValue = fabs(Matrixline[i*n + i]);
				}
			}
#pragma omp for
			for (int j = i + 1; j < n; j++) //����� ������������� ��������
			{
				float tmp = fabs(Matrixline[j*n + i]);
				if (tmp > maxValue)
				{
					maxN = j;
					maxValue = tmp;
				}
			}
#pragma omp single
			{
				if (maxN > i)
				{
					swaprow << <n, 1 >> >(dev_Matrix, i, maxN, n);
				}
				else if (maxValue == 0)
				{
					printf("null in matrix\n");
				}
			}
#pragma omp for
			for (int j = i + 1; j < n; j++)
			{
				proizrow << <n, th >> >(dev_Matrix, i, j, n);
			}
#pragma omp single
			{
				hipMemcpy(Matrixline, dev_Matrix, n*n*sizeof(float), hipMemcpyDeviceToHost);
			}
		}
	}
	end = omp_get_wtime();

	hipMemcpy(Matrixline, dev_Matrix, n*n*sizeof(float), hipMemcpyDeviceToHost);

	end = end - start;
	cu->timecudaver = end;

	if (n < 10)
	{
		for (int i = 0; i < n; i++)
		{
			for (int j = 0; j < n; j++)
			{
				printf("%.2f\t", Matrixline[i*n + j]);
			}
			printf("\n");
		}
	}

	return Matrix;
}