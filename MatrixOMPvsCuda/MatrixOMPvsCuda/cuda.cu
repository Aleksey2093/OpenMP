#include "hip/hip_runtime.h"
#include "Main.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>

__device__ void swaprow(double *Matrix, int row1, int row2, int n)
{
	int r = blockIdx.x*blockDim.x;
	int idx = r + threadIdx.x;
	if (idx >= row2*n && idx < (row2+1)*n)
	{
		int id = idx - row2*n + row1*n;
		int id2 = idx;//threadIdx.x + row2*n;
		//printf("id: %d, id2: %d",id,id2);
		double tmp;
		tmp = Matrix[id];
		Matrix[id] = Matrix[id2];
		Matrix[id2] = tmp;
	}
}

__device__ int maxN;
__device__ int *copy;
__device__ double maxValue;
__global__ void seamax(double *Matrix, int i, int n)
{
	if (blockIdx.x*blockDim.x + threadIdx.x == 0)
	{
		maxN = i;
		maxValue = fabs(Matrix[i*n + i]);
		for (int j = i + 1; j < n; j++) //����� ������������� ��������
		{
			double tmp = fabs(Matrix[j*n + i]);
			if (tmp > maxValue)
			{
				maxN = j;
				maxValue = tmp;
			}
		}
		copy = &maxN;
	}
	__syncthreads();
	if (maxN > i)
	{
		swaprow(Matrix, i, maxN, n);
	}
	else if (maxValue == 0)
	{
		if (blockIdx.x*blockDim.x + threadIdx.x == 0)
			printf("nulllll please stop \n");
	}
}

__global__ void proizrow(double *Matrix, int i, int j, int n)
{
	int id2 = blockIdx.x*blockDim.x + threadIdx.x; //�����������
	if (id2 < n*n)
	{
		int id = i*n + threadIdx.x; //������ ������
		int jni = j*n + i;
		double jim = Matrix[jni] / Matrix[i*n + i];
		__syncthreads();
		if (id2 == jni)
		{
			Matrix[id2] = 0;
		}
		else if (id2 > jni && id2 < (-1)*jni - i + n)
		{
			Matrix[id2] = Matrix[id2] - Matrix[id] * (jim);
		}
	}
}

double** Main::getMatrixFromCuda(double** Matrix, Main *cu)
{
	int n = SIZE;
	double end, start;
	double *dev_Matrix, *Matrixline = new double[n*n];
	int g = (n*n + 127) / 128, th = 128;
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			Matrixline[i*n + j] = Matrix[i][j];
		}
	}
	hipMalloc((void**)&dev_Matrix, n*n*sizeof(double));
	hipMemcpy(dev_Matrix, Matrixline, n*n*sizeof(double), hipMemcpyHostToDevice);
	
	if (n < 10)
	{
		printf("\n");
		for (int i = 0; i < n; i++)
		{
			for (int j = 0; j < n; j++)
			{
				printf("%.2f\t", Matrixline[i*n + j]);
			}
			printf("\n");
		}
		printf("\n");
	}
	dim3 grid(n, (n+127) / 128);
	start = omp_get_wtime();
	for (int i = 0; i < n-1; i++)
	{
		seamax << <n, th >> >(dev_Matrix, i, n);
#pragma omp parallel for
		for (int j = i + 1; j < n; j++)
		{
			proizrow << <n, th >> >(dev_Matrix, i, j, n);
		}
	}
	end = omp_get_wtime();
	hipMemcpy(Matrixline, dev_Matrix, n*n*sizeof(double), hipMemcpyDeviceToHost);
	hipFree(dev_Matrix);
	end = end - start;
	cu->timecudaver = end;

	if (n < 10)
	{
		for (int i = 0; i < n; i++)
		{
			for (int j = 0; j < n; j++)
			{
				printf("%.2f\t", Matrixline[i*n + j]);
			}
			printf("\n");
		}
	}

	return Matrix;
}